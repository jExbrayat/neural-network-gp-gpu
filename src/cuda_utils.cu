#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda_utils.cuh"

CudaThrowError::CudaThrowError(hipError_t error): error(error) {}
void CudaThrowError::throwError(std::string custom_msg) {
    if (error != hipSuccess) {
        std::cerr << custom_msg << hipGetErrorString(error) << std::endl;
    }
}

CudaMatrixMemory::CudaMatrixMemory(const int rows, const int cols) : rows(rows), cols(cols) {
    size_t free, total;
    printf("\n");
    hipMemGetInfo(&free,&total);   
    printf("%d MB free of total %d MB\n",free/1024/1024,total/1024/1024);
  
    memory_size = sizeof(float) * rows * cols;
    hipError_t err = hipMalloc((void**)&device_ptr, memory_size);
    CudaThrowError throwErr(err);
    throwErr.throwError("hipMalloc failed: ");

    if (err == hipSuccess) {
        std::cout << "Allocated " << memory_size /1024/1024<< "MB of memory"<< std::endl;
    }   
}

CudaMatrixMemory::~CudaMatrixMemory() {
    if (device_ptr) {
        hipFree(device_ptr);
        std::cout << "Freeing cuda memory" << std::endl;
    }
}

void CudaMatrixMemory::sendMatrix2Device(const float *carray) {
    hipError_t err = hipMemcpy(device_ptr, carray, memory_size, hipMemcpyHostToDevice);
    CudaThrowError throwErr(err);
    throwErr.throwError("hipMemcpy failed: ");

    std::cout << "Performed hipMemcpy" << std::endl; 
}

/**
 * @brief Allocate host memory into host_ptr and perform hipMemcpy from device to host.
 * The user need to free the allocated memory in returned host_ptr !
 */
float* CudaMatrixMemory::allocAndSend2Host() {
    // Allocate memory for the host
    float* host_ptr = new float[rows * cols]; // Use new[] for proper cleanup with delete[]
    
    if (host_ptr == nullptr) { // Check for successful allocation
        throw std::runtime_error("Memory allocation failed on host.");
    }
    
    // Copy data from device to host
    hipError_t err = hipMemcpy(host_ptr, device_ptr, memory_size, hipMemcpyDeviceToHost);
    CudaThrowError throwErr(err);
    throwErr.throwError("hipMemcpy failed: ");

    std::cout << "Performed hipMemcpy" << std::endl; 

    return host_ptr;
}

void CudaGrid::setKernelGrid(const int blocksize_x, const int blocksize_y, const int rows, const int cols) {
    threads = dim3(blocksize_x, blocksize_y);
    grid = dim3((cols + blocksize_x - 1) / blocksize_x, (rows + blocksize_y - 1) / blocksize_y);
}